
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *a, int *b, int *c) {
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx];
}

int main() {
    const int arraySize = 5;
    const int arrayBytes = arraySize * sizeof(int);

    int h_a[arraySize] = {1, 2, 3, 4, 5};
    int h_b[arraySize] = {10, 20, 30, 40, 50};
    int h_c[arraySize];

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, arrayBytes);
    hipMalloc((void**)&d_b, arrayBytes);
    hipMalloc((void**)&d_c, arrayBytes);

    hipMemcpy(d_a, h_a, arrayBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arrayBytes, hipMemcpyHostToDevice);

    add<<<1, arraySize>>>(d_a, d_b, d_c);

    hipMemcpy(h_c, d_c, arrayBytes, hipMemcpyDeviceToHost);

    for (int i = 0; i < arraySize; i++) {
        std::cout << h_c[i] << " ";
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
